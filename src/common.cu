#include "common.cuh"
#include "logging.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        __logCritical("CUDA ASSERT FAILURE :: %s %s %d", hipGetErrorString(code), file, line);

        if (abort)
            exit(code);
    }
}

void gpuErrorCheck(const char *file, int line)
{
    // --- Errors
    auto errSync = hipGetLastError();
    auto errAsync = hipDeviceSynchronize();

    // --- Status
    bool success = true;
    std::stringstream ss;

    // --- Error Check : synchronous
    if (errSync != hipSuccess)
    {
        success = false;
        __logCritical("Synchronous Kernel Error :: %s", hipGetErrorString(errSync));
        ss << "CUDA Error :: " << hipGetErrorString(errSync);
    }

    // --- Error Check : asynchronous
    if (errAsync != hipSuccess)
    {
        success = false;
        __logCritical("Synchronous Kernel Error :: %s", hipGetErrorString(errAsync));
        ss << "CUDA Error Async :: " << hipGetErrorString(errAsync);
    }

    if(not success)
        throw std::runtime_error(ss.str());
}

void* strict_malloc(size_t mem_size, const char* filename, int line)
{
    void* host_ptr = malloc(mem_size);

    // --- Error Checking
    if (host_ptr == nullptr) {
        __logCritical("MALLOC FAILURE :: Allocating %zu bytes of memory", mem_size);
        exit(EXIT_FAILURE);
    }

    return host_ptr;
}
